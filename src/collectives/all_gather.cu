/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "collectives.h"

ncclResult_t ncclAllGatherFunc(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  size_t nbytes = count*ncclTypeSize(datatype);
  INFO(COLL,"opCount %lx sendbuff %p recvbuff %p count %zi size %zi datatype %d op %d comm %p [nranks=%d] stream %p", comm->opCount, sendbuff, recvbuff, count, nbytes, datatype, op, comm, comm->nRanks, stream);
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, nbytes, hipMemcpyDeviceToDevice, stream));
  } else {
    NCCLCHECK(transportSaveProxies(ALLGATHER_SUBSTEPS, ALLGATHER_BUFCHUNKS, comm->nRanks-1, comm->nRanks, nbytes*comm->nRanks, proxyPatternRing, comm));
    NCCLCHECK(saveKernel(ncclCollAllGather, sendbuff, recvbuff, nbytes, ncclInt8, op, root, comm, stream, nbytes*comm->nRanks, 1));
  }
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclAllGather, const void* sendbuff, void* recvbuff, size_t sendcount,
    ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllGather(const void* sendbuff, void* recvbuff, size_t sendcount,
    ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream) {
  return ncclEnqueueCheck(ncclAllGatherFunc, "AllGather", sendbuff, recvbuff, sendcount, datatype,
          ncclSum, 0, comm, stream);
}
